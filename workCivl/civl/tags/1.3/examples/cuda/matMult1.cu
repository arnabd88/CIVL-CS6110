// TODO add assertions to check equivalence of implementation results
#ifdef _CIVL
#include <civlc.cvh>
#endif
/***********************************************************************
* FILENAME:  MM.cu
*            Matrix Multiplication
*            Matrix operands have row-major order.
*
* C = A * B
* Multiplies two square matrices (NxN * NxN).
* Matrix values have type double.
*
* A simple CUDA program has a basic workflow:
*     1)  Initialize matrix operands as double-precision arrays on host (CPU).
*     2)  Copy operands from host memory to GPU memory.
*     3)  Apply matrix operaton to operands on GPU
*     4)  Copy result from GPU memory to host memory.
*
*
* CUDA C Programming Guide Version 4.2 (3.2.3, p.22):
* http://developer.download.nvidia.com/compute/DevZone/docs/html/C/doc/CUDA_C_Programming_Guide.pdf
*
* MM with linearized matrix operands:
* http://www.hpcwire.com/hpcwire/2008-10-08/compilers_and_more_programming_gpus_today.html
*
*************************************************************************/
// online source: https://www.rcac.purdue.edu/userinfo/resources/carter/compile/MM.cu

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h" 

#ifdef _CIVL
$input int N;
$input int TILE_WIDTH;
#else
#define N 1024               /* size of square matrix                   */
#define TILE_WIDTH 16
#endif

#ifdef _CIVL
    $input double A[N*N];
    $input double B[N*N];
#endif

/* MM kernel using global (not shared) memory.                          */
__global__
void myMM_global (const double * const A, const double * const B, double *C, int width) {

    /* Get row and column from block and thread IDs                     */
    int row = (blockDim.y*blockIdx.y) + threadIdx.y;
    int col = (blockDim.x*blockIdx.x) + threadIdx.x;

    /* Initialize result of one element which one thread computes.      */
    double result=0.0;

    /* Compute one element of the matrix product.                       */
    for (int i = 0; i < width; ++i)
        result += A[row*width + i] * B[i*width + col];

    /* Store the result of one matrix element in matrix C.              */
    C[row * width + col] = result;
}


/* MM kernel using shared memory.                                       */
__global__
void myMM_shared (const double * const A, const double * const B, double* C, int width) {
    __shared__ double A_shared[TILE_WIDTH][TILE_WIDTH];
    __shared__ double B_shared[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    /* Identify the row and column of the C element to work on.         */
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    double result = 0.0;

    /* Loop over the A and B tiles required to compute the C element.   */
    for (int phase = 0; phase < width/TILE_WIDTH; ++phase) {
        /* Shared effort: loading of A and B tiles into shared memory.  */
        A_shared[ty][tx] = A[row*width + (phase*TILE_WIDTH + tx)];
        B_shared[ty][tx] = B[col + (phase*TILE_WIDTH + ty)*width]; 
        __syncthreads(); 

        for (int k = 0; k < TILE_WIDTH; ++k)
            result += A_shared[ty][k] * B_shared[k][tx];
        __syncthreads();

    }
    C[row*width+col] = result;
}


/************************************************************************/
/************************************************************************/
/************************************************************************/ 
 

int main (int argc, char** argv) {

#ifdef _CIVL
    $assume(argc == 2);
    $assume(atoi(argv[1]) == 0);
#endif

    /* Set device based on input from command line            */
    if (argc > 1) {
        if (hipSetDevice(atoi(argv[1])) != hipSuccess) {
            int num_devices;
            hipGetDeviceCount(&num_devices);
            fprintf(stderr, "Error initializing device %s,\
 device value must be 0-%d\n", argv[1], (num_devices-1));
            return 0;
        }
    } else {
        fprintf(stderr, "Usage: %s gpu_device\n", argv[0]);
        return 0;
    }

    /* Declare CPU arrays.                                              */
#ifndef _CIVL
    double A[N*N],B[N*N];
#endif
    double C[N*N];       /* linearized CPU double arrays  */ 
    int r,c;

    /* Declare GPU arrays.                                              */
    double *G_A,*G_B,*G_C;             /* linearized GPU double arrays  */
	size_t size_a,size_b,size_c;       /* size of linearized array in bytes */
    size_a = size_b = size_c = N*N;

    /* Setup a clock.                                                   */
    hipEvent_t start, stop;
    float CPU_elapsedtime, GPU_global_elapsedtime, GPU_shared_elapsedtime;
    hipEventCreate(&start);
    hipEventCreate(&stop);




    /* 1)  Initialize matrix operands as double-precision arrays on host (CPU). */
#ifndef _CIVL
    for (r=0;r<N;++r)
    for (c=0;c<N;++c) {
        A[r*N+c] = 1.0;
        B[r*N+c] = 1.0;
    }
#endif


/*-----------------------------------------------------------------------*/

    /* MM on a CPU.                                                      */
    hipEventRecord(start,0);
    for (int r = 0; r < N; ++r )
    for (int c = 0; c < N; ++c )
    for (int k = 0; k < N; ++k )
        C[r*N+c] += A[r*N+c] * B[k*N+c];
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&CPU_elapsedtime,start,stop);
    printf("                                                            speedup\n");
    printf("                                                            -------\n");
    printf("Elapsed time in CPU:                   %7.1f milliseconds\n", CPU_elapsedtime); 
/*-----------------------------------------------------------------------*/

    /* MM on Global Memory of GPGPU.                                     */
    hipEventRecord(start,0);

    /* 2)  Copy operands from CPU memory to GPGPU memory.                */
    hipMalloc((void**)&G_A,size_a*sizeof(double));  /* alloc A in GPGPU */
    hipMalloc((void**)&G_B,size_b*sizeof(double));  /* alloc B in GPGPU */
    hipMalloc((void**)&G_C,size_c*sizeof(double));  /* alloc C in GPGPU */
    hipMemcpy(G_A,A,size_a*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(G_B,B,size_b*sizeof(double),hipMemcpyHostToDevice);

    /* 3)  Apply matrix operation to operands on GPGPU                   */
    /*     There is no partial final block in this example.              */
    dim3 block = {TILE_WIDTH,TILE_WIDTH,1};      /* using a 2D block: 16,16,1 */
    dim3 grid = {N/TILE_WIDTH,N/TILE_WIDTH,1};   /* as many 16x16-thread blocks as needed: */
    myMM_global<<< grid,block >>>(G_A,G_B,G_C,N);  /* grid(16,16,1)  */ 

    /* 4)  Copy result from GPGPU memory to CPU memory.                  */
    hipMemcpy(C,G_C,size_c*sizeof(double),hipMemcpyDeviceToHost);

    /* Deallocate memory on GPGPU.                                       */
    hipFree(G_A);
    hipFree(G_B);
    hipFree(G_C);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&GPU_global_elapsedtime,start,stop);
    printf("Elapsed time in GPU (global memory):   %7.1f milliseconds  %5.1f\n",
           GPU_global_elapsedtime,CPU_elapsedtime/GPU_global_elapsedtime);
//*
    printf("\nGLOBAL MEMORY:\n");
    for (r=0;r<N;++r)
    for (c=0;c<N;++c) {
        printf("%2d,%2d   %g\n", r,c,C[r*N+c]);
	}
//*/
/*-----------------------------------------------------------------------*/

    /* MM on Shared Memory of GPGPU.                                     */
    hipEventRecord(start,0);

    /* 2)  Copy operands from CPU memory to GPGPU memory.                */
    hipMalloc((void**)&G_A,size_a*sizeof(double));  /* alloc A in GPGPU */
    hipMalloc((void**)&G_B,size_b*sizeof(double));  /* alloc B in GPGPU */
    hipMalloc((void**)&G_C,size_c*sizeof(double));  /* alloc C in GPGPU */
    hipMemcpy(G_A,A,size_a*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(G_B,B,size_b*sizeof(double),hipMemcpyHostToDevice);

    /* 3)  Apply matrix operation to operands on GPGPU                   */
    /*     There is not partial final block in this example.             */
    /*     Use the same grid and block from the previous case.           */
    myMM_shared<<< grid,block >>>(G_A,G_B,G_C,N);

    /* 4)  Copy result from GPGPU memory to CPU memory.                  */
    hipMemcpy(C,G_C,size_c*sizeof(double),hipMemcpyDeviceToHost);

    /* Deallocate memory on GPGPU.                                       */
    hipFree(G_A);
    hipFree(G_B);
    hipFree(G_C);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&GPU_shared_elapsedtime,start,stop);
    printf("Elapsed time in GPU (shared memory):   %7.1f milliseconds  %5.1f\n",
           GPU_shared_elapsedtime,CPU_elapsedtime/GPU_shared_elapsedtime);
//*
    printf("\nSHARED MEMORY:\n");
    for (r=0;r<N;++r)
    for (c=0;c<N;++c) {
        printf("%2d,%2d   %g\n", r,c,C[r*N+c]);
	}
//*/
/*-----------------------------------------------------------------------*/ 

    /* Deallocate the clock.                                             */
    hipEventDestroy(start);
    hipEventDestroy(stop); 

	return 0; 
}

