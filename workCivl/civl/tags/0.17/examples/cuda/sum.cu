/**
* This is an example from the paper "Formal Semantics of Heterogeneous CUDA-C: 
* A Modular Approach with Applications" by Chris Hathhorn et al. 
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#ifdef _CIVL
$input int N;
$input int N_B;
$assume 1 <= N && N <= N_B;
$input int NBLOCKS;
$input int NBLOCKS_B;
$assume 1 <= NBLOCKS && NBLOCKS <= NBLOCKS_B;
$assume NBLOCKS <= N;
$assume N % NBLOCKS == 0;
$assume N % 2 == 0;
$assume NBLOCKS % 2 == 0;
#else
#define N 8
#define NBLOCKS 4
#endif
#define NTHREADS (N/NBLOCKS)

__global__ void sum(int* in, int* out) {
  extern __shared__ int shared[];
  int i, tid = threadIdx.x,
         bid = blockIdx.x,
         bdim = blockDim.x;
         
  shared[tid] = in[bid * bdim + tid];
  
  __syncthreads();
  if(tid < bdim/2) {
    shared[tid] += shared[bdim/2 + tid];
  }
  __syncthreads();
  if(tid == 0) {
    for (i = 1; i != (bdim/2) + (bdim%2); ++i) {
      shared[0] += shared[i];
    }
    out[bid] = shared[0];
  }
}

int main(void) {

  int i, *dev_in, *dev_out, host[N];
#ifdef _CIVL
  int seqSum = 0;
#endif
  
  printf("INPUT: ");
  for(i = 0; i != N; ++i) {
    host[i] = (21*i + 29) % 100;
#ifdef _CIVL
    seqSum += host[i];
#endif
    printf(" %d ", host[i]);
  }
  printf("\n");
  
  hipMalloc(&dev_in, N * sizeof(int));
  hipMalloc(&dev_out, NBLOCKS * sizeof(int));
  
  hipMemcpy(dev_in, host, N * sizeof(int),
         hipMemcpyHostToDevice);
  sum<<<NBLOCKS, NTHREADS, NTHREADS * sizeof(int)>>>(
         dev_in, dev_out);
  sum<<<1, NBLOCKS, NBLOCKS * sizeof(int)>>>(
         dev_out, dev_out);
  hipMemcpy(host, dev_out, sizeof(int),
         hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  printf("OUTPUT: %u\n", *host);
#ifdef _CIVL
  $assert *host == seqSum;
#endif

  hipFree(dev_in);
  hipFree(dev_out);
  return 0;
}

