#include "hip/hip_runtime.h"
/**
* This is an example from the paper "Formal Semantics of Heterogeneous CUDA-C: 
* A Modular Approach with Applications" by Chris Hathhorn et al. 
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 8
#define NBLOCKS 4
#define NTHREADS (N/NBLOCKS)

__global__ void sum(int* in, int* out) {
  extern __shared__ int shared[];
  int i, tid = threadIdx.x,
         bid = blockIdx.x,
         bdim = blockDim.x;
         
  shared[tid] = in[bid * bdim + tid];
  
  __syncthreads();
  if(tid < bdim/2) {
    shared[tid] += shared[bdim/2 + tid];
  }
  __syncthreads();
  if(tid == 0) {
    for (i = 1; i != (bdim/2) + (bdim%2); ++i) {
      shared[0] += shared[i];
    }
    out[bid] = shared[0];
  }
}

int main(void) {
  int i, *dev_in, *dev_out, host[N];
  
  printf("INPUT: ");
  for(i = 0; i != N; ++i) {
    host[i] = (21*i + 29) % 100;
    printf(" %d ", host[i]);
  }
  printf("\n");
  
  hipMalloc(&dev_in, N * sizeof(int));
  hipMalloc(&dev_out, NBLOCKS * sizeof(int));
  
  hipMemcpy(dev_in, host, N * size(int),
         hipMemcpyHostToDevice);
  sum<<<NBLOCKS, NTHREADS, NTHREADS * sizeof(int)>>>(
         dev_in, dev_out);
  sum<<<1, NBLOCKS, NBLOCKS * sizeof(int)>>>(
         dev_out, dev_out);
  hipMemcpy(host, dev_out, sizeof(int),
         hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  
  printf("OUTPUT: %u\n", *host);
  hipFree(dev_in);
  hipFree(dev_out);
  return 0;
}

