#include "hip/hip_runtime.h"
#ifdef _CIVL
#include <civlc.cvh>
#endif
//http://www.arc.vt.edu/resources/software/cuda/docs/cuda-omp.cu

#include <omp.h>
#include <hip/hip_runtime.h>
#include <stdio.h>      
#include <stdlib.h>      


#ifdef _CIVL
$input int BLOCKS;
$input int BLOCK_B;
$assume(1 <= BLOCKS && BLOCKS <= BLOCK_B);
$input int THREADS_PER_BLOCK;
$input int THREADS_B;
$assume(1 <= THREADS_PER_BLOCK && THREADS_PER_BLOCK <= THREADS_B);
#else
#define BLOCKS 64
#define THREADS_PER_BLOCK 128
#endif

// A kernel that increments each array element by the value b

__global__ void kernelAddConstant(int *g_a, const int b)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	g_a[idx] += b;
}

// Check whether each element was incremented by the value b
int correctResult(int *data, const int n, const int b)
{
	for(int i = 0; i < n; i++)
        	if(data[i] != i + b)
                	return 0;
	return 1;
}

int main(int argc, char *argv[])
{
	// Variable which holds number of GPUs
	int num_gpus = 0;   

	// Determine the number of CUDA capable GPUs
        hipGetDeviceCount(&num_gpus);
        if(num_gpus < 1)
        {
                printf("No CUDA Capable GPU(s) Detected \n");
                return 1;
        }

        // Display the CPU and GPU processor specification         
	int num_procs = omp_get_num_procs();
	printf("number of host CPUs:\t%d\n", num_procs);
    	printf("number of CUDA devices:\t%d\n", num_gpus);
	for(int i = 0; i < num_gpus; i++)
    	{
        	hipDeviceProp_t dprop;
        	hipGetDeviceProperties(&dprop, i);
                printf("\t Device %d is a %s\n", i, dprop.name);
    	}


	// Initialize the variables 
    	unsigned int n = num_gpus * THREADS_PER_BLOCK * BLOCKS;
    	unsigned int nbytes = n * sizeof(int);
        int *a = 0;             // pointer to data on the CPU
        int b = 3;              // value by which each array array element will be incremented
        a = (int*)malloc(nbytes);
        
	if(0 == a)
        {
                printf("couldn't allocate CPU memory\n");
                return 1;
        }
        
	for(unsigned int i = 0; i < n; i++)
        	a[i] = i;
    
	// Set the number of threads to the number of GPUs on the system
	omp_set_num_threads(num_gpus);

	#pragma omp parallel
    	{
        	unsigned int cpu_thread_id = omp_get_thread_num();
                unsigned int num_cpu_threads = omp_get_num_threads();

                // Assign and check the GPU device for each thread
                int gpu_id = -1;
                hipSetDevice(cpu_thread_id % num_gpus);        
                hipGetDevice(&gpu_id);

                printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id, num_cpu_threads, gpu_id);

		// Variable on the device associated with this CPU thread
                int *d_a = 0; 

		// Variable for the CPU
                int *sub_a = a + cpu_thread_id * n / num_cpu_threads;
   
                unsigned int nbytes_per_kernel = nbytes / num_cpu_threads;
                dim3 gpu_threads = {THREADS_PER_BLOCK, 1, 1};  // 128 threads per block
                dim3 gpu_blocks = {(n / (gpu_threads.x * num_cpu_threads)), 1, 1};

		//Allocate memory on the device
          	hipMalloc((void**)&d_a, nbytes_per_kernel);

		//Initialize the array on the device with zeros
          	hipMemset(d_a, 0, nbytes_per_kernel);

		//Copy data from host to device
	       	hipMemcpy(d_a, sub_a, nbytes_per_kernel, hipMemcpyHostToDevice);
	
		//Launch the kernel
        	kernelAddConstant<<<gpu_blocks, gpu_threads>>>(d_a, b);

		//Copy the result  from the device to the host
          	hipMemcpy(sub_a, d_a, nbytes_per_kernel, hipMemcpyDeviceToHost);
          
		//Deallocate the memory on the device
		hipFree(d_a);

	}
        

        if(hipSuccess != hipGetLastError()) {
		int err_num = hipGetLastError();
		const char * err_str = hipGetErrorString(err_num);
		printf("%s\n", err_str);
	}


	//Check for correctness of the result
    	if(correctResult(a, n, b)) {
#ifdef _CIVL
		$assert(($true));
#endif
        	printf("Test PASSED\n");
    	} else
        	printf("Test FAILED\n");

	//Deallocate the CPU memory 
	free(a);    

	// deprecated
    	// hipDeviceReset();

    	return 0;
}

